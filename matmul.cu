#include "hip/hip_runtime.h"
#include<vector>
#include "matrix_utils.h"


struct ctlInput{
    int inner_product_size; // The size of A_rows or B_cols to take the sum sum over
    int out_rows;
    int out_cols;
};

__global__ void matmul_GPU(ctlInput *control, float *A_arr, float *B_arr, float *output){
    /* Simple matmul routine for a single GPU core.
       Assumes that input tensors have the correct dimensions for a matrix multiplication.
       Assumes output tensor has the correct dimension for a matrix multiplication. 
    */ 

    int i = threadIdx.x;
    int j = threadIdx.y;

    int inner_product_size = control->inner_product_size;
    int output_dims[2] = {control->out_rows, control->out_cols};

    float subsum = 0;
    for(int k=0; k<inner_product_size; k++){
        float A_row = get_linear_index(output_dims, i, k);
        float B_col = get_linear_index(output_dims, k, j);
        subsum += A_row * B_col;
    
    }
    int idx_lin = get_linear_index(output_dims, i, j);
    output[idx_lin] = subsum;
}

std::vector<std::vector<float>> matmul_cuda(std::vector<std::vector<float>> A, std::vector<std::vector<float>> B){
   
    is_well_formed(A); is_well_formed(B); 
    check_compatibility(A, B);
    
    auto A_dims = get_dims(A);
    auto B_dims = get_dims(B);

    auto A_flattened = flatten_2D_matrix(A);
    auto B_flattened = flatten_2D_matrix(B);

    // Copy ctl structure and input matrices into linear buffer
    ctlInput control = {A_flattened.second, B_flattened.second, A_dims.second*B_dims.first};
    ctlInput *control_gpu;
    size_t mem_ctlIn = sizeof(ctlInput);
    hipMalloc(&control_gpu, mem_ctlIn);
    hipMemcpy(&control_gpu, &control, mem_ctlIn, hipMemcpyHostToDevice);
    
    // Create linear buffer for matrix A
    float *A_arr;
    size_t mem_A = sizeof(float) * A_flattened.second;
    hipMalloc(&A_arr, mem_A);
    hipMemcpy(A_arr, A_flattened.first, mem_A, hipMemcpyHostToDevice);

    // Create linear buffer for matrix B
    float *B_arr;
    size_t mem_B = sizeof(float) * B_flattened.second;
    hipMalloc(&B_arr, mem_B);
    hipMemcpy(B_arr, B_flattened.first, mem_B, hipMemcpyHostToDevice);

    // Create output memory buffer on GPU
    float *output_arr;
    size_t mem_c = sizeof(float) * A_dims.second*B_dims.first;
    hipMalloc(&output_arr, mem_c);

    int num_blocks = 1;
    dim3 threads_per_block(A_dims.second, B_dims.first);
    matmul_GPU<<<num_blocks, threads_per_block>>>(control_gpu, A_arr, B_arr, output_arr);

    // Get computed matrix off GPU
    auto C_arr = (float *)malloc(mem_c);
    hipMemcpy(C_arr, &output_arr, mem_c, hipMemcpyDeviceToHost);
    std::pair<float *, int> C_flattened = std::make_pair(C_arr, mem_c);
    auto C_dims = std::make_pair(A_dims.second, B_dims.first);

    std::vector<std::vector<float>> C = unflatten_1D_array(C_flattened, C_dims);

    return C;

}

